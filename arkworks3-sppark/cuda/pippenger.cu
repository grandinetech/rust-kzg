// Copyright Supranational LLC
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0

#include <hip/hip_runtime.h>

#include <ff/bls12-381.hpp>

#include <ec/jacobian_t.hpp>
#include <ec/xyzz_t.hpp>

using namespace bls12_381;

typedef jacobian_t<fp_t> point_t;
typedef xyzz_t<fp_t> bucket_t;
typedef bucket_t::affine_inf_t affine_t;
typedef fr_t scalar_t;

#include <msm/pippenger.cuh>

#ifndef __CUDA_ARCH__

extern "C"
void *prepare_msm(const affine_t points[], size_t npoints, size_t ffi_affine_sz) {
    return new msm_t<bucket_t, point_t, affine_t, scalar_t>{points, npoints, ffi_affine_sz};
}

extern "C"
RustError mult_pippenger_prepared(void *msm, point_t* out, size_t npoints, const scalar_t scalars[]) {
    return static_cast<msm_t<bucket_t, point_t, affine_t, scalar_t>*>(msm)->invoke(*out, slice_t<scalar_t>{scalars, npoints}, true);
}

extern "C"
RustError mult_pippenger(point_t* out, const affine_t points[], size_t npoints,
                                       const scalar_t scalars[], size_t ffi_affine_sz)
{
    return mult_pippenger<bucket_t>(out, points, npoints, scalars, true, ffi_affine_sz);
}
#endif